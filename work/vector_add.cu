#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void vectorAdd(int*A, int*B,int *C, int N) {
    int i = blockDim.x*blockIdx.x+threadIdx.x;
    if (i < N){
        C[i] = A[i] + B[i];
    }
}

int main(){
    int N = 1 << 20;
    size_t sizeN = sizeof(int)*N;
    int* A = (int*)malloc(sizeN);
    int* B = (int*)malloc(sizeN);
    int* C = (int*)malloc(sizeN);

    for (int i=0;i<N;i++){
        A[i] = i;
        B[i] = i+2;
    }
    int* d_A, *d_B, *d_C;

    hipMalloc((void**)&d_A,sizeN);
    hipMalloc((void**)&d_B,sizeN);
    hipMalloc((void**)&d_C,sizeN);

    hipMemcpy(d_A,A,sizeN,hipMemcpyHostToDevice);

    hipMemcpy(d_B,B,sizeN,hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (N+threadsPerBlock -1 )/threadsPerBlock;

    vectorAdd<<<threadsPerBlock,blocksPerGrid>>>(d_A,d_B,d_C,N);

    hipMemcpy(C,d_C,sizeN,hipMemcpyDeviceToHost);


    for (int i=0;i<10;i++){
        printf("Key: %d, Value: %d",i,C[i]);
    }

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(A);
    free(B);
    free(C);
    return 0;


}