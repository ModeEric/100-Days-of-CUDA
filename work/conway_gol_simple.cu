#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>
#include <ctime>

#define GRID_WIDTH 32
#define GRID_HEIGHT 32

__global__ void conway_game_simple(unsigned char* current, unsigned char* next){
    int x = blockDim.x*blockIdx.x + threadIdx.x;
    int y = blockDim.y*blockIdx.y + threadIdx.y;
    if (x<GRID_WIDTH && y<GRID_HEIGHT){
        int liveNeighbors = 0;
        for(int i=-1;i<=1;i+=1){
            for(int j=-1; j<=1;j+=1){
                int nx = x+i;
                int ny = y+j;
                if (nx<GRID_WIDTH && ny<GRID_HEIGHT && nx>=0 && ny>=0){
                    liveNeighbors+=current[ny*GRID_WIDTH+nx];
                }
            }
        }
        char cell = current[y*GRID_WIDTH+x];
        liveNeighbors-=cell;

        if(cell==1 && (liveNeighbors==2 || liveNeighbors==3)){
            next[y*GRID_WIDTH+x]= 1;
        }
        else{
            next[y*GRID_WIDTH+x]=0;
        }
        if(cell==0 && liveNeighbors==3){
            next[y*GRID_WIDTH+x]=1;
        }
    }
}


int main(){
    int sizeN =sizeof(char)*GRID_HEIGHT*GRID_WIDTH;
    unsigned char* grid = (unsigned char*)malloc(sizeN);
    unsigned char* current = (unsigned char*)malloc(sizeN);
    unsigned char* grid_C;
    unsigned char* current_C;

    hipMalloc((void**)&grid_C,sizeN);
    hipMalloc((void**)&current_C,sizeN);

    for(int i=0;i<GRID_HEIGHT;i++){
        for (int j=0; j<GRID_WIDTH;j++){
            grid[i*GRID_WIDTH+j] = 0;
            current[i*GRID_WIDTH+j] = 0;
        }
    }
    hipMemcpy(grid_C,grid,sizeN,hipMemcpyHostToDevice);
    hipMemcpy(current_C,current,sizeN,hipMemcpyHostToDevice);

    dim3 threadsPerBlock(16,16);
    dim3 numBlocks((GRID_WIDTH+threadsPerBlock.x-1)/threadsPerBlock.x,(GRID_WIDTH+threadsPerBlock.y-1)/threadsPerBlock.y);
    conway_game_simple<<<numBlocks,threadsPerBlock>>>(current_C,grid_C);
    hipError_t cudErr = hipGetLastError();
    if (cudErr!=hipSuccess){
        printf("Cuda error: %s",cudErr)
    }
    hipDeviceSynchronize();

    hipMemcpy(grid,grid_C,sizeN,hipMemcpyDeviceToHost);
    for(int i=0;i <10; i++){
        for(int j=0;j<10;j++){
            printf("%d ",grid[i*GRID_WIDTH+j]);
        }
        printf("\n");
    }
    printf("\n");
    hipFree(current_C);
    hipFree(grid_C);
    free(grid);
    free(current);
    return 0;
}